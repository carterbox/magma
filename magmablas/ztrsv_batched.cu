#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @author Tingxing Dong
       @author Azzam Haidar

       @precisions normal z -> s d c
*/

#include "magma_internal.h"
#include "magma_templates.h"
#include "batched_kernel_param.h"

#define PRECISION_z

#include "gemm_template_device_defs.cuh"
#include "trsv_template_device.cuh"
#include "trsv_template_kernel_batched.cuh"

#define NB 256  //NB is the 1st level blocking in recursive blocking, BLOCK_SIZE is the 2ed level, NB=256, BLOCK_SIZE=64 is optimal for batched

#define NUM_THREADS 128 //64 //128

#define BLOCK_SIZE_N 128
#define DIM_X_N 128
#define DIM_Y_N 1

#define BLOCK_SIZE_T 32
#define DIM_X_T 16
#define DIM_Y_T 8

#include "ztrsv_template_device.cuh"

#define A(i, j)  (A + (i) + (j)*lda)   // A(i, j) means at i row, j column


/******************************************************************************/
static void
magmablas_ztrsv_small_batched(
        magma_uplo_t uplo, magma_trans_t transA, magma_diag_t diag,
        magma_int_t n,
        magmaDoubleComplex **dA_array, magma_int_t Ai, magma_int_t Aj, magma_int_t ldda,
        magmaDoubleComplex **dx_array, magma_int_t xi, magma_int_t incx,
        magma_int_t batchCount, magma_queue_t queue )
{
    if     ( n <=  2 )
        trsv_small_batched<magmaDoubleComplex,  2>(uplo, transA, diag, n, dA_array, ldda, dx_array, incx, Ai, Aj, xi, batchCount, queue );
    else if( n <=  4 )
        trsv_small_batched<magmaDoubleComplex,  4>(uplo, transA, diag, n, dA_array, ldda, dx_array, incx, Ai, Aj, xi, batchCount, queue );
    else if( n <=  8 )
        trsv_small_batched<magmaDoubleComplex,  8>(uplo, transA, diag, n, dA_array, ldda, dx_array, incx, Ai, Aj, xi, batchCount, queue );
    else if( n <= 16 )
        trsv_small_batched<magmaDoubleComplex, 16>(uplo, transA, diag, n, dA_array, ldda, dx_array, incx, Ai, Aj, xi, batchCount, queue );
    else if( n <= 32 )
        trsv_small_batched<magmaDoubleComplex, 32>(uplo, transA, diag, n, dA_array, ldda, dx_array, incx, Ai, Aj, xi, batchCount, queue );
    else
        printf("error in function %s: nrowA must be less than 32\n", __func__);
}

/******************************************************************************/
static magma_int_t magma_get_ztrsv_batched_nb(magma_int_t n)
{
    if      ( n > 2048 ) return 2048;
    else if ( n > 1024 ) return 1024;
    else if ( n >  512 ) return 512;
    else if ( n >  256 ) return 256;
    else if ( n >  128 ) return 128;
    else if ( n >   64 ) return  64;
    else if ( n >   32 ) return  32;
    else if ( n >   16 ) return  16;
    else if ( n >    8 ) return   8;
    else if ( n >    4 ) return   4;
    else if ( n >    2 ) return   2;
    else return 1;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
extern "C" void
magmablas_ztrsv_recursive_batched(
        magma_uplo_t uplo, magma_trans_t transA, magma_diag_t diag,
        magma_int_t n,
        magmaDoubleComplex **dA_array, magma_int_t Ai, magma_int_t Aj, magma_int_t ldda,
        magmaDoubleComplex **dx_array, magma_int_t xi, magma_int_t incx,
        magma_int_t batchCount, magma_queue_t queue )
{
#define dA_array(i,j) dA_array, i, j
#define dx_array(i)   dx_array, i

    const magmaDoubleComplex c_one    = MAGMA_Z_ONE;
    const magmaDoubleComplex c_negone = MAGMA_Z_NEG_ONE;

    magma_int_t shape = -1;
    if      (transA == MagmaNoTrans  && uplo == MagmaLower) { shape = 0; } // NL
    else if (transA == MagmaNoTrans  && uplo == MagmaUpper) { shape = 1; } // NU
    else if (transA != MagmaNoTrans  && uplo == MagmaLower) { shape = 2; } // TL | CL
    else if (transA != MagmaNoTrans  && uplo == MagmaUpper) { shape = 3; } // TU | CU

    // stopping condition
    if(n <= 32){
        magmablas_ztrsv_small_batched(
            uplo, transA, diag, n,
            dA_array(Ai, Aj), ldda,
            dx_array(xi), incx, batchCount, queue );
        return;
    }

    const int n2 = magma_get_ztrsv_batched_nb(n);
    const int n1 = n - n2;


//#define DBG
#ifdef DBG
    printf("n1 = %d, n2 = %d\n", n1, n2);
    magmaDoubleComplex* tmpx = NULL;
    magma_getvector(1, sizeof(magmaDoubleComplex*), dx_array, 1, &tmpx, 1, queue);
#endif

    switch(shape) {
        case 0: // Nl
        {

            magmablas_ztrsv_recursive_batched(
                uplo, transA, diag, n1,
                dA_array(Ai, Aj), ldda,
                dx_array(xi    ), incx,
                batchCount, queue );

            #ifdef DBG
            magma_zprint_gpu(n, 1, tmpx, n, queue);
            #endif

            magmablas_zgemv_batched_core(
                transA, n2, n1,
                c_negone, dA_array(Ai+n1, Aj), ldda,
                          dx_array(xi       ), incx,
                c_one,    dx_array(xi+n1    ), incx,
                batchCount, queue );
            #ifdef DBG
            magma_zprint_gpu(n, 1, tmpx, n, queue);
            #endif

            magmablas_ztrsv_recursive_batched(
                uplo, transA, diag, n2,
                dA_array(Ai+n1, Aj+n1), ldda,
                dx_array(xi+n1       ), incx,
                batchCount, queue );
            #ifdef DBG
            magma_zprint_gpu(n, 1, tmpx, n, queue);
            #endif
        }
        break;
        ////////////////////////////////////////////////////////////////////////
        case 1: // NU
        {
            magmablas_ztrsv_recursive_batched(
                uplo, transA, diag, n2,
                dA_array(Ai+n1, Aj+n1), ldda,
                dx_array(xi+n1       ), incx,
                batchCount, queue );

            magmablas_zgemv_batched_core(
                transA, n1, n2,
                c_negone, dA_array(Ai, Aj+n1), ldda,
                          dx_array(xi+n1    ), incx,
                c_one,    dx_array(xi       ), incx,
                batchCount, queue );

            magmablas_ztrsv_recursive_batched(
                uplo, transA, diag, n1,
                dA_array(Ai, Aj), ldda,
                dx_array(xi    ), incx,
                batchCount, queue );
        }
        break;
        ////////////////////////////////////////////////////////////////////////
        case 2: // TL || CL
        {
            magmablas_ztrsv_recursive_batched(
                uplo, transA, diag, n2,
                dA_array(Ai+n1, Aj+n1), ldda,
                dx_array(xi+n1       ), incx,
                batchCount, queue );

            magmablas_zgemv_batched_core(
                transA, n2, n1,
                c_negone, dA_array(Ai+n1, Aj), ldda,
                          dx_array(xi+n1    ), incx,
                c_one,    dx_array(xi       ), incx,
                batchCount, queue );


            magmablas_ztrsv_recursive_batched(
                uplo, transA, diag, n1,
                dA_array(Ai, Aj), ldda,
                dx_array(xi    ), incx,
                batchCount, queue );
        }
        break;
        ////////////////////////////////////////////////////////////////////////
        case 3: // TU | lCU
        {
            magmablas_ztrsv_recursive_batched(
                uplo, transA, diag, n1,
                dA_array(Ai, Aj), ldda,
                dx_array(xi    ), incx,
                batchCount, queue );

            magmablas_zgemv_batched_core(
                transA, n1, n2,
                c_negone, dA_array(Ai, Aj+n1), ldda,
                          dx_array(xi       ), incx,
                c_one,    dx_array(xi+n1    ), incx,
                batchCount, queue );

            magmablas_ztrsv_recursive_batched(
                uplo, transA, diag, n2,
                dA_array(Ai+n1, Aj+n1), ldda,
                dx_array(xi+n1       ), incx,
                batchCount, queue );
        }
        break;
        ////////////////////////////////////////////////////////////////////////
        default:; // propose something
    }
#undef dA_array
#undef dx_array
}


/******************************************************************************/
template< const int BLOCK_SIZE, const int DIM_X, const int DIM_Y,  const int TILE_SIZE, const int flag, const magma_uplo_t uplo, const magma_trans_t trans, const magma_diag_t diag>
__global__ void
ztrsv_notrans_kernel_outplace_batched(
    int n,
    magmaDoubleComplex **A_array, int lda,
    magmaDoubleComplex **b_array, int incb,
    magmaDoubleComplex **x_array)
{
    int batchid = blockIdx.z;

    ztrsv_notrans_device<BLOCK_SIZE, DIM_X, DIM_Y, TILE_SIZE, flag, uplo, trans, diag>(n, A_array[batchid], lda, b_array[batchid], incb, x_array[batchid]);
}


/******************************************************************************/
template<const int BLOCK_SIZE, const int DIM_X, const int DIM_Y,  const int TILE_SIZE, const int flag, const magma_uplo_t uplo, const magma_trans_t trans, const magma_diag_t diag>
__global__ void
ztrsv_trans_kernel_outplace_batched(
    int n,
    magmaDoubleComplex **A_array, int lda,
    magmaDoubleComplex **b_array, int incb,
    magmaDoubleComplex **x_array)
{
    int batchid = blockIdx.z;
    ztrsv_trans_device<BLOCK_SIZE, DIM_X, DIM_Y, TILE_SIZE, flag, uplo, trans, diag>(n, A_array[batchid], lda, b_array[batchid], incb, x_array[batchid]);
}

/******************************************************************************/
// notrans driver
template< const int BLOCK_SIZE, const int DIM_X, const int DIM_Y,  const int TILE_SIZE, const int flag, const magma_uplo_t uplo, const magma_trans_t trans, const magma_diag_t diag>
void
ztrsv_notrans_outplace_batched(
    int n,
    magmaDoubleComplex **A_array, int lda,
    magmaDoubleComplex **b_array, int incb,
    magmaDoubleComplex **x_array,
    magma_int_t batchCount, magma_queue_t queue)
{
    magma_int_t max_batchCount = queue->get_maxBatch();
    size_t shmem = n * sizeof(magmaDoubleComplex);
    dim3 threads( NUM_THREADS, 1, 1 );

    for(magma_int_t i = 0; i < batchCount; i+=max_batchCount) {
        magma_int_t ibatch = min(max_batchCount, batchCount-i);
        dim3 grid( 1, 1, ibatch );

        ztrsv_notrans_kernel_outplace_batched<BLOCK_SIZE, DIM_X, DIM_Y, TILE_SIZE, flag, uplo, trans, diag>
        <<<grid, threads, shmem, queue->hip_stream()>>>
        (n, A_array+i, lda, b_array+i, incb, x_array+i);
    }
}

/******************************************************************************/
// trans driver
template<const int BLOCK_SIZE, const int DIM_X, const int DIM_Y,  const int TILE_SIZE, const int flag, const magma_uplo_t uplo, const magma_trans_t trans, const magma_diag_t diag>
void
ztrsv_trans_outplace_batched(
    int n,
    magmaDoubleComplex **A_array, int lda,
    magmaDoubleComplex **b_array, int incb,
    magmaDoubleComplex **x_array,
    magma_int_t batchCount, magma_queue_t queue)
{
    magma_int_t max_batchCount = queue->get_maxBatch();
    size_t shmem = n * sizeof(magmaDoubleComplex);
    dim3 threads( NUM_THREADS, 1, 1 );

    for(magma_int_t i = 0; i < batchCount; i+=max_batchCount) {
        magma_int_t ibatch = min(max_batchCount, batchCount-i);
        dim3 grid( 1, 1, ibatch );

        ztrsv_trans_kernel_outplace_batched<BLOCK_SIZE, DIM_X, DIM_Y, TILE_SIZE, flag, uplo, trans, diag>
        <<<grid, threads, shmem, queue->hip_stream()>>>
        (n, A_array+i, lda, b_array+i, incb, x_array+i);
    }
}


/******************************************************************************/
extern "C" void
magmablas_ztrsv_outofplace_batched(
    magma_uplo_t uplo, magma_trans_t trans, magma_diag_t diag,
    magma_int_t n,
    magmaDoubleComplex ** A_array, magma_int_t lda,
    magmaDoubleComplex **b_array, magma_int_t incb,
    magmaDoubleComplex **x_array,
    magma_int_t batchCount, magma_queue_t queue,
    magma_int_t flag)
{
    /* Check arguments */
    magma_int_t info = 0;
    if ( uplo != MagmaUpper && uplo != MagmaLower ) {
        info = -1;
    } else if ( trans != MagmaNoTrans && trans != MagmaTrans && trans != MagmaConjTrans ) {
        info = -2;
    } else if ( diag != MagmaUnit && diag != MagmaNonUnit ) {
        info = -3;
    } else if (n < 0) {
        info = -5;
    } else if (lda < max(1,n)) {
        info = -8;
    }

    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;
    }


    // quick return if possible.
    if (n == 0)
        return;

    if (trans == MagmaNoTrans) {
        if (uplo == MagmaUpper) {
            if (diag == MagmaNonUnit) {
                if (flag == 0) {
                    ztrsv_notrans_outplace_batched< BLOCK_SIZE_N, DIM_X_N, DIM_Y_N, MagmaBigTileSize, 0, MagmaUpper, MagmaNoTrans, MagmaNonUnit >
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
                else {
                    ztrsv_notrans_outplace_batched< BLOCK_SIZE_N, DIM_X_N, DIM_Y_N, MagmaBigTileSize, 1, MagmaUpper, MagmaNoTrans, MagmaNonUnit >
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
            }
            else if (diag == MagmaUnit) {
                if (flag == 0) {
                    ztrsv_notrans_outplace_batched< BLOCK_SIZE_N, DIM_X_N, DIM_Y_N, MagmaBigTileSize, 0, MagmaUpper, MagmaNoTrans, MagmaUnit >
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
                else {
                    ztrsv_notrans_outplace_batched< BLOCK_SIZE_N, DIM_X_N, DIM_Y_N, MagmaBigTileSize, 1, MagmaUpper, MagmaNoTrans, MagmaUnit >
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
            }
        }
        else { //Lower
            if (diag == MagmaNonUnit) {
                if (flag == 0) {
                    ztrsv_notrans_outplace_batched< BLOCK_SIZE_N, DIM_X_N, DIM_Y_N, MagmaBigTileSize, 0, MagmaLower, MagmaNoTrans, MagmaNonUnit >
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
                else {
                    ztrsv_notrans_outplace_batched< BLOCK_SIZE_N, DIM_X_N, DIM_Y_N, MagmaBigTileSize, 1, MagmaLower, MagmaNoTrans, MagmaNonUnit >
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
            }
            else if (diag == MagmaUnit) {
                if (flag == 0) {
                    ztrsv_notrans_outplace_batched< BLOCK_SIZE_N, DIM_X_N, DIM_Y_N, MagmaBigTileSize, 0, MagmaLower, MagmaNoTrans, MagmaUnit>
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
                else {
                    ztrsv_notrans_outplace_batched< BLOCK_SIZE_N, DIM_X_N, DIM_Y_N, MagmaBigTileSize, 1, MagmaLower, MagmaNoTrans, MagmaUnit>
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
            }
        }
    }
    else if (trans == MagmaTrans) {
        if (uplo == MagmaUpper) {
            if (diag == MagmaNonUnit) {
                if (flag == 0) {
                    ztrsv_trans_outplace_batched< BLOCK_SIZE_T, DIM_X_T, DIM_Y_T, MagmaBigTileSize, 0, MagmaUpper, MagmaTrans, MagmaNonUnit >
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
                else {
                    ztrsv_trans_outplace_batched< BLOCK_SIZE_T, DIM_X_T, DIM_Y_T, MagmaBigTileSize, 1, MagmaUpper, MagmaTrans, MagmaNonUnit >
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
            }
            else if (diag == MagmaUnit) {
                if (flag == 0) {
                    ztrsv_trans_outplace_batched< BLOCK_SIZE_T, DIM_X_T, DIM_Y_T, MagmaBigTileSize, 0, MagmaUpper, MagmaTrans, MagmaUnit >
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
                else {
                    ztrsv_trans_outplace_batched< BLOCK_SIZE_T, DIM_X_T, DIM_Y_T, MagmaBigTileSize, 1, MagmaUpper, MagmaTrans, MagmaUnit >
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
            }
        }
        else {
            if (diag == MagmaNonUnit) {
                if (flag == 0) {
                    ztrsv_trans_outplace_batched< BLOCK_SIZE_T, DIM_X_T, DIM_Y_T,MagmaBigTileSize, 0, MagmaLower, MagmaTrans, MagmaNonUnit >
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
                else {
                    ztrsv_trans_outplace_batched< BLOCK_SIZE_T, DIM_X_T, DIM_Y_T, MagmaBigTileSize, 1, MagmaLower, MagmaTrans, MagmaNonUnit >
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
            }
            else if (diag == MagmaUnit) {
                if (flag == 0) {
                    ztrsv_trans_outplace_batched< BLOCK_SIZE_T, DIM_X_T, DIM_Y_T,MagmaBigTileSize, 0, MagmaLower, MagmaTrans, MagmaUnit >
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
                else {
                    ztrsv_trans_outplace_batched< BLOCK_SIZE_T, DIM_X_T, DIM_Y_T, MagmaBigTileSize, 1, MagmaLower, MagmaTrans, MagmaUnit >
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
            }
        }
    }
    else if (trans == MagmaConjTrans) {
        if (uplo == MagmaUpper) {
            if (diag == MagmaNonUnit) {
                if (flag == 0) {
                    ztrsv_trans_outplace_batched< BLOCK_SIZE_T, DIM_X_T, DIM_Y_T, MagmaBigTileSize, 0, MagmaUpper, MagmaConjTrans, MagmaNonUnit >
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
                else {
                    ztrsv_trans_outplace_batched< BLOCK_SIZE_T, DIM_X_T, DIM_Y_T, MagmaBigTileSize, 1, MagmaUpper, MagmaConjTrans, MagmaNonUnit >
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
            }
            else if (diag == MagmaUnit) {
                if (flag == 0) {
                    ztrsv_trans_outplace_batched< BLOCK_SIZE_T, DIM_X_T, DIM_Y_T, MagmaBigTileSize, 0, MagmaUpper, MagmaConjTrans, MagmaUnit >
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
                else {
                    ztrsv_trans_outplace_batched< BLOCK_SIZE_T, DIM_X_T, DIM_Y_T, MagmaBigTileSize, 1, MagmaUpper, MagmaConjTrans, MagmaUnit >
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
            }
        }
        else {
            if (diag == MagmaNonUnit) {
                if (flag == 0) {
                    ztrsv_trans_outplace_batched< BLOCK_SIZE_T, DIM_X_T, DIM_Y_T,MagmaBigTileSize, 0, MagmaLower, MagmaConjTrans, MagmaNonUnit >
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
                else {
                    ztrsv_trans_outplace_batched< BLOCK_SIZE_T, DIM_X_T, DIM_Y_T, MagmaBigTileSize, 1, MagmaLower, MagmaConjTrans, MagmaNonUnit >
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
            }
            else if (diag == MagmaUnit) {
                if (flag == 0) {
                    ztrsv_trans_outplace_batched< BLOCK_SIZE_T, DIM_X_T, DIM_Y_T,MagmaBigTileSize, 0, MagmaLower, MagmaConjTrans, MagmaUnit >
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
                else {
                    ztrsv_trans_outplace_batched< BLOCK_SIZE_T, DIM_X_T, DIM_Y_T, MagmaBigTileSize, 1, MagmaLower, MagmaConjTrans, MagmaUnit >
                    (n, A_array, lda, b_array, incb, x_array, batchCount, queue);
                }
            }
        }
    }
}


/******************************************************************************/
extern "C" void
magmablas_ztrsv_recursive_outofplace_batched(
    magma_uplo_t uplo, magma_trans_t trans, magma_diag_t diag,
    magma_int_t n,
    magmaDoubleComplex **A_array, magma_int_t lda,
    magmaDoubleComplex **b_array, magma_int_t incb,
    magmaDoubleComplex **x_array,
    magma_int_t batchCount, magma_queue_t queue)
{
#define dW0_displ_ (const magmaDoubleComplex**) dW0_displ
#define dW1_displ_ (const magmaDoubleComplex**) dW1_displ

    /* Check arguments */
    magma_int_t info = 0;
    if ( uplo != MagmaUpper && uplo != MagmaLower ) {
        info = -1;
    } else if ( trans != MagmaNoTrans && trans != MagmaTrans && trans != MagmaConjTrans ) {
        info = -2;
    } else if ( diag != MagmaUnit && diag != MagmaNonUnit ) {
        info = -3;
    } else if (n < 0) {
        info = -5;
    } else if (lda < max(1,n)) {
        info = -8;
    }

    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;
    }

    // quick return if possible.
    if (n == 0)
        return;


    //Init x_array with zero
    //magmablas_zlaset_batched(MagmaFull, n, incb, MAGMA_Z_ZERO, MAGMA_Z_ZERO, x_array, n, batchCount, queue);

    //memory allocation takes 0.32ms

    magmaDoubleComplex **dW0_displ  = NULL;
    magmaDoubleComplex **dW1_displ  = NULL;
    magmaDoubleComplex **dW2_displ  = NULL;

    magma_int_t alloc = 0;

    alloc += magma_malloc((void**)&dW0_displ,  batchCount * sizeof(*dW0_displ));
    alloc += magma_malloc((void**)&dW1_displ,  batchCount * sizeof(*dW1_displ));
    alloc += magma_malloc((void**)&dW2_displ,  batchCount * sizeof(*dW2_displ));

    if (alloc != 0)
    {
        magma_free( dW0_displ );
        magma_free( dW1_displ );
        magma_free( dW2_displ );

        info = MAGMA_ERR_DEVICE_ALLOC;
        return;
    }

    magma_int_t col = n;

    if (trans == MagmaNoTrans)
    {
        for (magma_int_t i=0; i < n; i+= NB)
        {
            magma_int_t jb = min(NB, n-i);

            if (uplo == MagmaUpper)
            {
                col -= jb;

                magma_zdisplace_pointers(dW0_displ, A_array, lda, col, col+jb, batchCount, queue);
                magma_zdisplace_pointers(dW1_displ, x_array, 1, col+jb, 0,     batchCount, queue);
                magma_zdisplace_pointers(dW2_displ, x_array, 1, col,    0,     batchCount, queue);
            }
            else
            {
                col = i;

                magma_zdisplace_pointers(dW0_displ, A_array, lda, col, 0, batchCount, queue);
                magma_zdisplace_pointers(dW1_displ, x_array, 1,   0,   0, batchCount, queue);
                magma_zdisplace_pointers(dW2_displ, x_array, 1,   col, 0, batchCount, queue);
            }

            //assume x_array contains zero elements
            magmablas_zgemv_batched(MagmaNoTrans, jb, i, MAGMA_Z_ONE, dW0_displ_, lda, dW1_displ_, 1, MAGMA_Z_ONE, dW2_displ, 1, batchCount, queue);

            magma_zdisplace_pointers(dW0_displ, A_array, lda,  col, col, batchCount, queue);
            magma_zdisplace_pointers(dW1_displ, b_array, 1, col*incb,   0, batchCount, queue);
            magma_zdisplace_pointers(dW2_displ, x_array, 1,    col,   0, batchCount, queue);

            magmablas_ztrsv_outofplace_batched(uplo, trans, diag,jb, dW0_displ, lda, dW1_displ, incb, dW2_displ, batchCount, queue, i);
        }
    }
    else
    {
        for (magma_int_t i=0; i < n; i += NB)
        {
            magma_int_t jb = min(NB, n-i);

            if (uplo == MagmaLower)
            {
                col -= jb;

                magma_zdisplace_pointers(dW0_displ, A_array, lda, col+jb, col, batchCount, queue);
                magma_zdisplace_pointers(dW1_displ, x_array, 1, col+jb, 0,     batchCount, queue);
                magma_zdisplace_pointers(dW2_displ, x_array, 1, col,    0,     batchCount, queue);
            }
            else
            {
                col = i;

                magma_zdisplace_pointers(dW0_displ, A_array, lda, 0, col,  batchCount, queue);
                magma_zdisplace_pointers(dW1_displ, x_array, 1,   0,   0, batchCount, queue);
                magma_zdisplace_pointers(dW2_displ, x_array, 1,   col, 0, batchCount, queue);
            }


            //assume x_array contains zero elements

            magmablas_zgemv_batched(trans, i, jb, MAGMA_Z_ONE, dW0_displ_, lda, dW1_displ_, 1, MAGMA_Z_ONE, dW2_displ, 1, batchCount, queue);

            magma_zdisplace_pointers(dW0_displ, A_array, lda,  col, col, batchCount, queue);
            magma_zdisplace_pointers(dW1_displ, b_array, 1, col*incb,   0, batchCount, queue);
            magma_zdisplace_pointers(dW2_displ, x_array, 1,    col,   0, batchCount, queue);

            magmablas_ztrsv_outofplace_batched(uplo, trans, diag, jb, dW0_displ, lda, dW1_displ, incb, dW2_displ, batchCount, queue, i);
        }
    }

    magma_free(dW0_displ);
    magma_free(dW1_displ);
    magma_free(dW2_displ);
}


/******************************************************************************/
extern "C" void
magmablas_ztrsv_work_batched(
    magma_uplo_t uplo, magma_trans_t trans, magma_diag_t diag,
    magma_int_t n,
    magmaDoubleComplex **A_array, magma_int_t lda,
    magmaDoubleComplex **b_array, magma_int_t incb,
    magmaDoubleComplex **x_array,
    magma_int_t batchCount, magma_queue_t queue)
{
    //magmablas_zlaset_batched(MagmaFull, n, incb, MAGMA_Z_ZERO, MAGMA_Z_ZERO, x_array, n, batchCount, queue);

    //magmablas_ztrsv_recursive_outofplace_batched

    magmablas_ztrsv_recursive_outofplace_batched(uplo, trans, diag, n, A_array, lda, b_array, incb, x_array, batchCount, queue);

    magmablas_zlacpy_batched( MagmaFull, n, incb, x_array, n, b_array, n, batchCount, queue);
}


/***************************************************************************//**
    Purpose
    -------
    ztrsv solves one of the matrix equations on gpu

        op(A)*x = b,   or
        x*op(A) = b,

    where alpha is a scalar, X and B are vectors, A is a unit, or
    non-unit, upper or lower triangular matrix and op(A) is one of

        op(A) = A,    or
        op(A) = A^T,  or
        op(A) = A^H.

    The vector x is overwritten on b.


    Arguments
    ----------

    @param[in]
    uplo    magma_uplo_t.
            On entry, uplo specifies whether the matrix A is an upper or
            lower triangular matrix as follows:
      -     = MagmaUpper:  A is an upper triangular matrix.
      -     = MagmaLower:  A is a  lower triangular matrix.

    @param[in]
    trans  magma_trans_t.
            On entry, trans specifies the form of op(A) to be used in
            the matrix multiplication as follows:
      -     = MagmaNoTrans:    op(A) = A.
      -     = MagmaTrans:      op(A) = A^T.
      -     = MagmaConjTrans:  op(A) = A^H.

    @param[in]
    diag    magma_diag_t.
            On entry, diag specifies whether or not A is unit triangular
            as follows:
      -     = MagmaUnit:     A is assumed to be unit triangular.
      -     = MagmaNonUnit:  A is not assumed to be unit triangular.

    @param[in]
    n       INTEGER.
            On entry, n N specifies the order of the matrix A. n >= 0.

    @param[in]
    A_array       Array of pointers, dimension (batchCount).
            Each is a COMPLEX_16 array A of dimension ( lda, n ),
            Before entry with uplo = MagmaUpper, the leading n by n
            upper triangular part of the array A must contain the upper
            triangular matrix and the strictly lower triangular part of
            A is not referenced.
            Before entry with uplo = MagmaLower, the leading n by n
            lower triangular part of the array A must contain the lower
            triangular matrix and the strictly upper triangular part of
            A is not referenced.
            Note that when diag = MagmaUnit, the diagonal elements of
            A are not referenced either, but are assumed to be unity.

    @param[in]
    lda     INTEGER.
            On entry, lda specifies the first dimension of A.
            lda >= max( 1, n ).

    @param[in]
    b_array     Array of pointers, dimension (batchCount).
            Each is a COMPLEX_16 array of dimension  n
            On exit, b is overwritten with the solution vector X.

    @param[in]
    incb    INTEGER.
            On entry,  incb specifies the increment for the elements of
            b. incb must not be zero.
            Unchanged on exit.

    @param[in]
    batchCount  INTEGER
                The number of matrices to operate on.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_trsv_batched
*******************************************************************************/
extern "C" void
magmablas_ztrsv_batched(
    magma_uplo_t uplo, magma_trans_t trans, magma_diag_t diag,
    magma_int_t n,
    magmaDoubleComplex **A_array, magma_int_t lda,
    magmaDoubleComplex **b_array, magma_int_t incb,
    magma_int_t batchCount,
    magma_queue_t queue)
{
    magma_int_t size_x = n * incb;

    magmaDoubleComplex *x=NULL;
    magmaDoubleComplex **x_array = NULL;

    magma_zmalloc( &x, size_x * batchCount);
    magma_malloc((void**)&x_array,  batchCount * sizeof(*x_array));

    magma_zset_pointer( x_array, x, n, 0, 0, size_x, batchCount, queue );

    magmablas_ztrsv_work_batched(uplo, trans, diag, n, A_array, lda, b_array, incb, x_array, batchCount, queue);

    magma_free(x);
    magma_free(x_array);
}
